#include "activations.cuh"

int main(){

    float x[5] = {0.0, 0.1, 0.2, 0.3, 0.4};
    float y[5] = {0.0, 0.0, 0.0, 0.0, 0.0};

    float* device_x;
    float* device_y;

    // Allocate device memory
    hipError_t err;
    err = hipMalloc(&device_x, 5*sizeof(float));
    if(err!=hipSuccess){
        printf("CUDA error: %s\n", hipGetErrorString(err));
        return -1;
    }

    err = hipMalloc(&device_y, 5*sizeof(float));
    if(err!=hipSuccess){
        printf("CUDA error: %s\n", hipGetErrorString(err));
        hipFree(device_x);
        return -1;
    }

    // Copy data from host to device
    err = hipMemcpy(device_x, x, 5*sizeof(float), hipMemcpyHostToDevice);
    if(err!=hipSuccess){
        printf("CUDA error: %s\n", hipGetErrorString(err));
        hipFree(device_x);
        hipFree(device_y);
        return -1;
    }

    // Launch kernel
    gelu_forward1(device_x, device_y, 5, 32);

    // Copy result from device to host
    err = hipMemcpy(y, device_y, 5*sizeof(float), hipMemcpyDeviceToHost);
    if(err!=hipSuccess){
        printf("CUDA error: %s\n", hipGetErrorString(err));
        hipFree(device_x);
        hipFree(device_y);
        return -1;
    }

    // Free device memory
    hipFree(device_x);
    hipFree(device_y);

    // Print results
    for(int i=0; i<5;++i){
        printf("result: %f\n", y[i]);
    }

    return 0;
}