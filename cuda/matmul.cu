#include "hip/hip_runtime.h"
#include "matmul.cuh"


// -----------------------------------------------------------------------------------------
// GPU kernels

__global__ void matmul_forward_kernel1(float* x, float* y, float* weight, float* bias,
                                       int B_in_r, int in_c, int ou_c){
    
    int B_in_r_dim = blockIdx.x * blockDim.x + threadIdx.x;
    int ou_c_dim = blockIdx.y * blockDim.y + threadIdx.y;
    if(B_in_r_dim < B_in_r && ou_c_dim < ou_c){
        float val = (bias != NULL) ? bias[ou_c_dim] : 0.f;
        for(int ic = 0; ic<in_c; ++ic){
            val += x[B_in_r_dim*in_c + ic] * weight[ic*ou_c + ou_c_dim];
        }

        y[B_in_r_dim*ou_c + ou_c_dim] = val;
    }
}

__global__ void matmul_backward_kernel1(float* x, float* weight, float* dx, float* dweight, float* dbias,
                                        float* dy, int B_in_r, int in_c, int ou_c){
    
    int B_in_r_dim = blockIdx.x * blockDim.x + threadIdx.x;
    int ou_c_dim = blockIdx.y * blockDim.y + threadIdx.y;

    if(B_in_r_dim < B_in_r && ou_c_dim < ou_c){
        float grad = dy[B_in_r_dim*ou_c + ou_c_dim];

        if(dbias != NULL){
            atomicAdd(&dbias[ou_c_dim], grad);
        }

        for(int ic = 0; ic < in_c; ++ic){
            atomicAdd(&dx[B_in_r_dim*in_c + ic], grad * weight[ic*ou_c + ou_c_dim]);
            atomicAdd(&dweight[ic*ou_c + ou_c_dim], grad * x[B_in_r_dim*in_c + ic]);
        }
    }
}

// Helper function to extract a tensor at index t in sequence T of the original tensor.
// This is used for modified matmul function, which will be called before the classfication stage
// of ViT.
// 
// @param orig          linearized original input tensors [B, T, H]
// @param extracted     linearized extracted tensors, slice at index t [B, 1, H]
// @param B             number of batches
// @param T             sequence length (patch length + 1)
// @param H             hidden dimension size
// @param t             index t in the sequence T to be sliced
//
__global__ void slice_tensor_at_t_kernel(float* orig, float* extracted,
                                         int B, int T, int H, int t){
    int b_dim = blockIdx.x * blockDim.x + threadIdx.x;
    int h_dim = blockIdx.y * blockDim.y + threadIdx.y;
    if(b_dim < B && h_dim < H){
        extracted[b_dim*H + h_dim] = orig[b_dim*T*H + t*H + h_dim];
    }
    __syncthreads();
}

// -----------------------------------------------------------------------------------------
// kernel launcher

void matmul_forward1(float* x, float* y, float* weight, float* bias,
                     int B, int in_r, int in_c, int ou_c, const int sqrt_block_size){
    
    dim3 gridDim(ceil_div(B*in_r, sqrt_block_size), ceil_div(ou_c, sqrt_block_size));
    dim3 blockDim(sqrt_block_size, sqrt_block_size);
    matmul_forward_kernel1<<<gridDim, blockDim>>>(x, y, weight, bias, B*in_r, in_c, ou_c);
}

void matmul_backward1(float* x, float* weight, float* dx, float* dweight, float* dbias,
                      float* dy, int B, int in_r, int in_c, int ou_c, const int sqrt_block_size){
    
    dim3 gridDim(ceil_div(B*in_r, sqrt_block_size), ceil_div(ou_c, sqrt_block_size));
    dim3 blockDim(sqrt_block_size, sqrt_block_size);
    matmul_backward_kernel<<<gridDim, blockDim>>>(x, weight, dx, dweight, dbias, dy, B*in_r, in_c, ou_c);
}

void matmul_forward_with_slicing_at_t(float* x, float* y, float* weight, float* bias,
                                      int B, int T, int H, int NC, int t, const int sqrt_block_size){
    
    float* extracted_x;
    hipMalloc(&extracted_x, B*H*sizeof(float));

    dim3 slice_gridDim(ceil_div(B, sqrt_block_size), ceil_div(H, sqrt_block_size));
    dim3 matmul_gridDim(ceil_div(B, sqrt_block_size), ceil_div(NC, sqrt_block_size));
    dim3 blockDim(sqrt_block_size, sqrt_block_size);

    slice_tensor_at_t_kernel<<<slice_gridDim, blockDim>>>(x, extracted_x, B, T, H, t);
    matmul_forward_kernel1<<<matmul_gridDim, matmul_gridDim>>>(extracted_x, y, weight, bias, B, H, NC);

    cudaDeviceSynchronoize();
    hipFree(extracted_x);
}