#include "hip/hip_runtime.h"
#include "vit.cuh"

void ViT_init_common(ViTModel* model){

    // Common inits outside of the model weights.
    // memory lazily initialized in forward() call.
    model->acts_memory = NULL;
    model->inputs = NULL;
    model->targets = NULL;
    model->inputs_test = NULL;
    model->targets_test = NULL;

    model->batch_size = 0;
    model->curr_batch_idx = 0;
    model->mean_loss = 0.f;
    model->nImages = 0;
    model->training_mode = true;

    model->batch_size_test = 0;
    model->curr_batch_idx_test = 0;
    model->mean_loss_test = 0.f;
    model->nImages_test = 0;

    model->data_train = NULL;
    model->data_test = NULL;
    model->labels_train = NULL;
    model->labels_test = NULL;

    // Get the device ID
    hipGetDevice(&model->deviceId);

    // Get the device properties
    hipGetDeviceProperties(&model->deviceProp, model->deviceId);

    // CUDA block size.
    model->max_num_threads = model->deviceProp.maxThreadsPerBlock;
    model->sqrt_max_num_threads = static_cast<int>(std::sqrt(model->max_num_threads));
    model->cubert_max_num_threads = static_cast<int>(std::cbrt(model->max_num_threads));

    printf("[INFO] Current CUDA deivce: %s, max threads/block = %d, sqrt max = %d, cubert max = %d\n", 
            model->deviceProp.name, model->max_num_threads, model->sqrt_max_num_threads, model->cubert_max_num_threads);

}

void ViT_allocate_weights(ViTModel* model){

    // fill in all the parameter tensor dimensions and types.
    fill_in_parameter_sizes(model->param_sizes, model->param_sizeof, &(model->config));
    model->num_parameters = 0;
    model->num_parameters_bytes = 0;
    for(int i=0; i<NUM_PARAMETER_TENSORS; ++i){
        model->num_parameters += model->param_sizes[i];
        model->num_parameters_bytes += model->param_sizes[i] * model->param_sizeof[i];
    }

    // Create memory for model parameters on the deivce.
    assert(model->params_memory == nullptr);
    model->params_memory = malloc_and_point_parameters(&(model->params), model->param_sizes, model->param_sizeof);
}

void ViT_allocate_states(ViTModel* model, int B){
    printf("[INFO] Allocating %d MiB for parameter gradients.\n", (int)round(model->num_parameters * sizeof(floatX)/(1024*1024)));
    assert(model->params_grads_memory == nullptr);
    model->params_grads_memory = malloc_and_point_parameters(&(model->params_grads), model->param_sizes, model->param_sizeof);

    model->batch_size = B;
    int im_C = model->config.channels;
    int im_H = model->config.image_height;
    int im_W = model->config.image_width;

    // Allocate the space for activation tensors and activation gradient tensors.
    fill_in_activation_sizes(&(model->acts), model->acts_specs, B, &(model->config));
    model->acts_memory = malloc_and_point_activations(model->acts_specs);
    fill_in_activation_sizes(&(model->acts_grads), model->acts_grads_specs, B, &(model->config));
    model->acts_grads_memory = malloc_and_point_activations(model->acts_grads_specs);
    size_t num_act_bytes = 0;
    for (size_t i = 0; i < NUM_ACTIVATION_TENSORS; i++) {
        num_act_bytes += model->acts_specs[i].size * sizeof_dtype(model->acts_specs[i].type);
    }
    num_act_bytes *= 2;
    printf("[INFO] Allocating %zd MiB for activation/gradients tensors.\n", num_act_bytes/(1024*1024));

    // Create memory for cahcing inputs and targets
    cudaCheck(hipMalloc((void**)&model->inputs, B*im_C*im_H*im_W*sizeof(float)));
    cudaCheck(hipMalloc((void**)&model->targets, B*sizeof(float)));
    cudaCheck(hipMalloc((void**)&model->accumulated_mean_loss, sizeof(float)));
    cudaCheck(hipHostMalloc((void**)&model->cpu_loss, sizeof(float)));

    // AdamW optimizer parameters.
    printf("[INFO] Allocating %zu MiB for AdamW optimizer state m.\n", sizeof(float)>>20);
    printf("[INFO] Allocating %zu MiB for AdamW optimizer state v.\n", sizeof(float)>>20);
    assert(model->m_memory == nullptr);
    assert(model->v_memory == nullptr);
    cudaCheck(hipMalloc((void**)&model->m_memory, sizeof(float)));
    cudaCheck(hipMalloc((void**)&model->v_memory, sizeof(float)));

    // Memory usage info
    size_t free, total;
    cudaCheck(hipMemGetInfo(&free, &total));
    printf("[INFO] Device memory usage %zd MiB / %zd MiB.\n", (total-free)/(1024*1024), total/(1024*1024));
}

void ViT_forward(ViTModel* model, const float* inputs, const int* targets, size_t B){
    if(model->params_memory == NULL){
        printf("[ERROR] model was not initialized properly.\n");
        exit(EXIT_FAILURE);
    }

    // Number parameters
    int NC = model->config.num_classes;
    int NL = model->config.num_layers;
    int NH = model->config.num_attention_heads;
    int im_C = model->config.channels;
    int im_H = model->config.image_height;
    int im_W = model->config.image_width;
    int H = model->config.hidden_size;
    int P = model->config.patch_size;

    // Sanity check
    assert(im_W%P==0 && im_H%P==0);

    int NP = (im_W/P)*(im_H/P); // number of patches 
    int T = NP + 1; // sequence length (+1 corresponds to cls_token)

    // Validate B is not larger than the values used at initialization.
    // Smaller B is okay for inference only.
    if(B > model->batch_size){
        printf("[ERROR] Model got B=%d, Desired: (max) B=%d.\n", (int)B, model->batch_size);
        exit(EXIT_FAILURE);
    }

    // Copy inputs, targets to the model.
    cudaCheck(hipMemcpy(model->inputs, inputs, B*im_C*im_H*im_W*sizeof(float)));
    cudaCheck(hipMemcpy(model->targets, targets, B*sizeof(int)));

    // Forward pass
    ParameterTensors params = model->params;
    ActivationTensors acts = model->acts;

    // Patch embedding
    conv2d_forward1(model->inputs, params.patch_embd_kernel, params.patch_embd_bias, acts.patch_embd,
                    B, im_C, im_H, im_W, H, P, P, P, 0, model->cubert_max_num_threads);
    
    // Embedding = pos_embedding + cat(cls_token, patch_embedding)
    embeddings_forward1(acts.patch_embd, params.cls_token, params.pos_embd, acts.encoded,
                        B, NP, H, im_H/P, im_W/P, model->cubert_max_num_threads);
    
    // Attention block layers
    floatX* residual;
    for(int l=0; l<NL; ++l){
        residual = l == 0 ? acts.encoded : acts.resi_mlp + (l-1)*B*T*H;

        // get the pointers of the weights for the current layer
        floatX* l_ln1w = params.ln1w + l*H;
        floatX* l_ln1b = params.ln1b + l*H;
        floatX* l_qkvw = params.qkvw + l*H*3*H; 
        floatX* l_qkvb = params.qkvb + l*3*H;
        floatX* l_attn_projw = params.attn_projw + l*H*H;
        floatX* l_attn_projb = params.attn_projb + l*H; 
        floatX* l_ln2w = params.ln2w + l*H; 
        floatX* l_ln2b = params.ln2b + l*H;
        floatX* l_mlpw = params.mlpw + l*H*4*H; 
        floatX* l_mlpb = params.mlpb + l*4*H; 
        floatX* l_mlp_projw = params.mlp_projw + l*4*H*H; 
        floatX* l_mlp_projb = params.mlp_projb + l*H;

        // get the pointers of the activations for the current layer
        float* l_ln1_mean = acts.ln1_mean + l*B*T; 
        float* l_ln1_rstd = acts.ln1_rstd + l*B*T; 
        floatX* l_ln1 = acts.ln1 + l*B*T*H; 
        floatX* l_qkv = acts.qkv + l*B*T*3*H; 
        floatX* l_preattn = acts.preattn + l*B*NH*T*T; 
        floatX* l_attn = acts.attn + l*B*NH*T*T; 
        floatX* l_attn_y = acts.attn_y + l*B*T*H; 
        floatX* l_attn_proj = acts.attn_proj + l*B*T*H; 
        floatX* l_resi_attn = acts.resi_attn + l*B*T*H; 
        float* l_ln2_mean = acts.ln2_mean + l*B*T;
        float* l_ln2_rstd = acts.ln2_rstd + l*B*T;
        floatX* l_ln2 = acts.ln2 + l*B*T*H; 
        floatX* l_mlph = acts.mlph + l*B*T*4*H;
        floatX* l_mlph_gelu = acts.mlph_gelu + l*B*T*4*H; 
        floatX* l_mlp_proj = acts.mlp_proj + l*B*T*H;
        floatX* l_resi_mlp = acts.resi_mlp + l*B*T*H;

        // attention block forward pass
        layernorm_forward1(residual, l_ln1_mean, l_ln1_rstd, l_ln1w, l_ln1b, l_ln1, B, T, H, model->max_num_threads);
        matmul_forward1(l_ln1, l_qkv, l_qkvw, l_qkvb, B, T, H, 3*H, model->sqrt_max_num_threads);
        attention_forward1(l_qkv, l_preattn, l_attn, l_attn_y, B, T, H, NH, model->max_num_threads);
        matmul_forward1(l_attn_y, l_attn_proj, l_attn_projw, l_attn_projb, B, T, H, H, model->sqrt_max_num_threads);
        residual_forward1(l_attn_proj, residual, l_resi_attn, B*T*H, model->max_num_threads);
        layernorm_forward1(l_resi_attn, l_ln2_mean, l_ln2_rstd, l_ln2w, l_ln2b, l_ln2, B, T, H, model->max_num_threads);
        matmul_forward1(l_ln2, l_mlph, l_mlpw, l_mlpb, B, T, H, 4*H, model->sqrt_max_num_threads);
        gelu_forward1(l_mlph, l_mlph_gelu, B*T*4*H, model->max_num_threads);
        matmul_forward1(l_mlph_gelu, l_mlp_proj, l_mlp_projw, l_mlp_projb, B, T, 4*H, H, model->sqrt_max_num_threads);
        residual_forward1(l_mlp_proj, l_resi_attn, l_resi_mlp, B*T*H, model->max_num_threads);
    }
    residual = acts.resi_mlp + (NL-1)*B*T*H; // (B, T, H)

    // classifier
    // The first index in the sequence T, corresponding to cls_token is responsible for 
    // classification prediction.
    matmul_forward_with_slicing_at_t2(residual, acts.logits, params.clsw, params.clsb, B, T, H, NC, 0, model->sqrt_max_num_threads);
    softmax_forward1(acts.logits, acts.probs, B, NC, model->max_num_threads);
    crossentropy_forward1(acts.probs, targets, acts.losses, B, NC, model->max_num_threads);

    // loss metric calculation for the model.
    // calculating mean loss on device.
    *(model->accumulated_mean_loss) = 0.f;
    for(int b=0; b<B; ++b){
        *(model->accumulated_mean_loss) += acts.losses[b];
    }

    // mean loss to be copied to host at logging step (completion of forward/backward/update cycle for the total steps)

}

void ViT_backward(ViTModel* model){
    if(model->params_grads_memory == nullptr || model->acts_grads_memory == nullptr){
        fprintf(stderr, "[ERROR] Need to allocate gradients before backward pass call.\n");
        exit(EXIT_FAILURE);
    }

    // Number parameters
    int B = model->batch_size;
    int NC = model->config.num_classes;
    int NL = model->config.num_layers;
    int NH = model->config.num_attention_heads;
    int im_C = model->config.channels;
    int im_H = model->config.image_height;
    int im_W = model->config.image_width;
    int H = model->config.hidden_size;
    int P = model->config.patch_size;
    int NP = (im_W/P)*(im_H/P);
    int T = NP + 1;

    // Sanity check was performed during forward pass, so safe to skip

    // Backward pass
    ParameterTensors params = model->params;
    ParameterTensors param_grads = model->params_grads;
    ActivationTensors acts = model->acts;
    ActivationTensors acts_grads = model->acts_grads;

    // Start the chainrule by filling in dlosses with 1.f/B
    float dloss_mean = 1.f/B;
    for(int b=0; b<B; ++b) {acts_grads.losses[b] = dloss_mean;}

    crossentropy_softmax_backward1(acts.probs, model->targets, acts_grads.logits, acts_grads.losses, B, NC, model->max_num_threads);
    floatX* residual = acts.resi_mlp + (NL-1)*B*T*H; // (B, T, H)
    floatX* dresidual = acts_grads.resi_mlp + (NL-1)*B*T*H;

}