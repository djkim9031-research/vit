#include "hip/hip_runtime.h"
#include "vit.cuh"

void ViT_init_common(ViTModel* model){

    // Common inits outside of the model weights.
    // memory lazily initialized in forward() call.
    model->acts_memory = NULL;
    model->inputs = NULL;
    model->targets = NULL;
    model->inputs_test = NULL;
    model->targets_test = NULL;

    model->batch_size = 0;
    model->curr_batch_idx = 0;
    model->mean_loss = 0.f;
    model->nImages = 0;
    model->training_mode = true;

    model->batch_size_test = 0;
    model->curr_batch_idx_test = 0;
    model->mean_loss_test = 0.f;
    model->nImages_test = 0;

    model->data_train = NULL;
    model->data_test = NULL;
    model->labels_train = NULL;
    model->labels_test = NULL;

    // Get the device ID
    hipGetDevice(&model->deviceId);

    // Get the device properties
    hipGetDeviceProperties(&model->deviceProp, model->deviceId);

}

void ViT_allocate_weights(ViTModel* model){

    // fill in all the parameter tensor dimensions and types.
    fill_in_parameter_sizes(model->param_sizes, model->param_sizeof, &(model->config));
    model->num_parameters = 0;
    model->num_parameters_bytes = 0;
    for(int i=0; i<NUM_PARAMETER_TENSORS; ++i){
        model->num_parameters += model->param_sizes[i];
        model->num_parameters_bytes += model->param_sizes[i] * model->param_sizeof[i];
    }

    // Create memory for model parameters on the deivce.
    assert(model->params_memory == nullptr);
    model->params_memory = malloc_and_point_parameters(&(model->params), model->param_sizes, model->param_sizeof);
}

void ViT_allocate_states(ViTModel* model, int B){
    printf("Allocating %d MiB for parameter gradients.\n", (int)round(model->num_parameters * sizeof(floatX)/(1024*1024)));
    assert(model->params_grads_memory == nullptr);
    model->params_grads_memory = malloc_and_point_parameters(&(model->params_grads), model->param_sizes, model->param_sizeof);

    model->batch_size = B;
    int im_C = model->config.channels;
    int im_H = model->config.image_height;
    int im_W = model->config.image_width;

    // Allocate the space for activation tensors and activation gradient tensors.
    fill_in_activation_sizes(&(model->acts), model->acts_specs, B, &(model->config));
    model->acts_memory = malloc_and_point_activations(model->acts_specs);
    fill_in_activation_sizes(&(model->acts_grads), model->acts_grads_specs, B, &(model->config));
    model->acts_grads_memory = malloc_and_point_activations(model->acts_grads_specs);
    size_t num_act_bytes = 0;
    for (size_t i = 0; i < NUM_ACTIVATION_TENSORS; i++) {
        num_act_bytes += model->acts_specs[i].size * sizeof_dtype(model->acts_specs[i].type);
    }
    num_act_bytes *= 2;
    printf("Allocating %zd MiB for activation/gradients tensors.\n", num_act_bytes/(1024*1024));

    // Create memory for cahcing inputs and targets
    cudaCheck(hipMalloc((void**)&model->inputs, B*im_C*im_H*im_W*sizeof(float)));
    cudaCheck(hipMalloc((void**)&model->targets, B*sizeof(float)));
    cudaCheck(hipMalloc((void**)&model->accumulated_mean_loss, sizeof(float)));
    cudaCheck(hipHostMalloc((void**)&model->cpu_loss, sizeof(float)));

    // AdamW optimizer parameters.
    printf("Allocating %zu MiB for AdamW optimizer state m.\n", sizeof(float)>>20);
    printf("Allocating %zu MiB for AdamW optimizer state v.\n", sizeof(float)>>20);
    assert(model->m_memory == nullptr);
    assert(model->v_memory == nullptr);
    cudaCheck(hipMalloc((void**)&model->m_memory, sizeof(float)));
    cudaCheck(hipMalloc((void**)&model->v_memory, sizeof(float)));

    // Memory usage info
    size_t free, total;
    cudaCheck(hipMemGetInfo(&free, &total));
    printf("[INFO] device memory usage %zd MiB / %zd MiB.\n", (total-free)/(1024*1024), total/(1024*1024));
}